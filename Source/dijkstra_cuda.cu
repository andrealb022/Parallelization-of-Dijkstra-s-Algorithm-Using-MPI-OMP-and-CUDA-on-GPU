#include "hip/hip_runtime.h"
/*
 * Course: High Performance Computing 2023/2024
 *
 * Lecturer: Francesco Moscato	fmoscato@unisa.it
 *
 * Student :
 * Alberti Andrea	0622702370	a.alberti2@studenti.unisa.it
 *
 *
 * Copyright (C) 2023 - All Rights Reserved
 *
 * This file is part of DijkstraFinalProjectHPC
 *
 * DijkstraFinalProjectHPC   is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * DijkstraFinalProjectHPC   is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with DijkstraFinalProjectHPC . If not, see <http://www.gnu.org/licenses/>.
/**
 *                                  REQUIREMENTS OF THE ASSIGNMENT
 *
 * Student shall provide a parallel version of Dijkstra algorithm with both "OpenMP + MPI" and "OpenMP + Cuda" approaches,
 * comparing results with a known solution on single-processing node.
 * Results and differences shall be discussed for different inputs (type and size).
 * The parallel algorithm used in "OpenMP + MPI" solution could not be the same of the "OpenMP + CUDA" approach.
 *
 * @file dijkstra_cuda.cu
 * @copyright Copyright (c) 2023
 */

#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <time.h>
#include <sys/time.h>
#include <assert.h>
#include <omp.h>
#include "graph.c"
#include "utility.c"

#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/**
 * @brief Finds the index of the minimum distance in the distances array that has not been visited.
 *
 * @param distances   Array of distances.
 * @param marker      Array indicating whether a node has been visited.
 * @param array_size  Size of the arrays.
 * @return            Index of the minimum distance.
 */
int searchMinIndex(int* distances, int* marker, int array_size) {
    int j;
    int local_min_pair[2];
    local_min_pair[0]=MAXINT;
    local_min_pair[1]=-1;

    #pragma omp parallel for private(j) shared(local_min_pair)
        for (j = 0; j < array_size; j++) {
            if (!marker[j] && distances[j] < local_min_pair[0] && distances[j] != MAXINT) {
                #pragma omp critical
                {
                    if (distances[j] < local_min_pair[0]) {
                        local_min_pair[0] = distances[j];
                        local_min_pair[1] = j;
                    }
                }
            }
        }
    marker[local_min_pair[1]] = 1;
    return local_min_pair[1];
}

/**
 * @brief CUDA kernel to update distances based on the Dijkstra algorithm.
 *
 * @param graph         Adjacency matrix of the graph.
 * @param node_dist     Array of distances from the source vertex.
 * @param path          Array representing the predecessor shortest path from the source node.
 * @param visited       Array indicating whether a node has been visited.
 * @param source        Source node to update distances for Dijkstra's algorithm.
 * @param num_vertices  Number of vertices in the graph.
 */
__global__ void cuda_update_distance(int* graph, int* node_dist, int* path, int* visited, int source, int num_vertices) {
    int node = blockIdx.x * blockDim.x + threadIdx.x;
    if(node < num_vertices){
        visited[source] = 1;
        int edge = graph[source * num_vertices + node];
        int new_dist = node_dist[source] + edge;
        if((edge != MAXINT) && (edge != 0)){
            if ((visited[node] != 1) && (new_dist < node_dist[node])) {
                node_dist[node] = new_dist;
                path[node] = source;
            }
        }
    }
}

/**
 * @brief Main function implementing the OMP/CUDA Dijkstra algorithm.
 *
 * @param argc Number of command-line arguments.
 * @param argv Command-line arguments.
 * @return     0 on successful execution, 1 on failure.
 */
int main(int argc, char* argv[]) {

    if (argc != 5) {
    printf("Usage: %s <number_of_vertices> <graph_type> <seed> <optimization_type>\n", argv[0]);
    exit(1);
    }

    struct timeval all_start, all_end, start, end, total_time, graph_creation_time, dijkstra_time, allocation_time;
    gettimeofday(&all_start, NULL); // the variable all_start is used to measure the run time of the program

    int N,n_edges;  //number of vertices,edges
    N = atoi(argv[1]);

    int numThreadsPerBlock;      // blockSize
    int minGridSize;    // grid min size
    int numBlocks;      //gridSize

    if (N <= 0){ // check if the number of vertices in input is coherent
        printf("Number of vertices is wrong!\n");
        exit(1);
    }
    if(atoi(argv[2])<0 || atoi(argv[2])>3){ // check if the type of graph is coherent
        printf("Type of graph is wrong!\n");
        exit(1);
    }
    if(atoi(argv[4])<0 || atoi(argv[4])>3){ // check if the optimization is coherent
        printf("Optimization choice is wrong!\n");
        exit(1);
    }

    //calculate blocksize optimally
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &numThreadsPerBlock, (void*)cuda_update_distance, 0, N);

    // Calculate gridSize & blockSize based on the size of the problem
    numBlocks = (N + numThreadsPerBlock - 1) / numThreadsPerBlock;
    dim3 gridSize(numBlocks, 1, 1);
    dim3 blockSize(numThreadsPerBlock, 1, 1);

    int type = atoi(argv[2]); // type is the type of graph to be created (e.g. sparse, dense, etc.)
    int seed = atoi(argv[3]); // seed is the random seed generator used to guarantee the same graph in the tests

    //host vectors
    int* graph;
    int* host_distances;
    int* host_path;
    int* host_visited;

    //device vectors
    int* device_graph;
    int* device_distances;
    int* device_path;
    int* device_visited;

    //start the timer to calculate the graph creation time
    gettimeofday(&start, NULL);

    //host allocations
    graph = createGraph(N);
    n_edges = createGraphByType(type, N, graph, seed);
    assert(graph != NULL); // check if the graph is created correctly
    gettimeofday(&end, NULL); // stop the timer and calculate the graph creation time
    timersub(&end, &start, &graph_creation_time);

    host_distances = (int*)malloc(N * sizeof(int));
    host_path = (int*)malloc(N * sizeof(int));
    host_visited = (int*)malloc(N * sizeof(int));

    // arrays inizializations
    initializeArray(host_distances, N, MAXINT);              //all node distances are infinity
    initializeArray(host_path, N, SOURCE_VERTEX);    //parent nodes are SOURCE (no parents yet)
    initializeArray(host_visited, N, 0);              //no nodes have been visited
    host_distances[SOURCE_VERTEX] = 0;                     //start distance for SOURCE_VERTEX is 0;
    host_path[SOURCE_VERTEX]= -1;                  //start partent for SOURCE_VERTEX is -1;

    //start the timer to calculate gpu allocation time
    gettimeofday(&start, NULL);

    //device allocations
    CUDA_SAFE_CALL(hipMalloc((void**)&device_graph, N * N * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&device_distances, N * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&device_path, N * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&device_visited, N * sizeof(int)));
    //gpu source        cpu source      memory size     HtD or DtH
    CUDA_SAFE_CALL(hipMemcpy(device_graph, graph, N * N * sizeof(int), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(device_distances, host_distances, N * sizeof(int), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(device_path, host_path, N * sizeof(int), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(device_visited, host_visited, N * sizeof(int), hipMemcpyHostToDevice));

    gettimeofday(&end, NULL); // stop the timer and calculate device allocation time
    timersub(&end, &start, &allocation_time);

    //start the timer to calculate dijkstra execution time
    gettimeofday(&start, NULL);

    for (int i = 0; i < N; i++) {
        //step 1: search min
        int min = searchMinIndex(host_distances, host_visited, N);

        //step 2: update distance
        cuda_update_distance<<<gridSize,blockSize>>>(device_graph, device_distances, device_path, device_visited, min, N);

        // transfers the updated distance vector from device to host
        hipDeviceSynchronize();
        CUDA_SAFE_CALL(hipMemcpy(host_distances, device_distances, N * sizeof(int), hipMemcpyDeviceToHost));
    }
    // transfers the path vector from device to host
    hipDeviceSynchronize();
    CUDA_SAFE_CALL(hipMemcpy(host_path, device_path, N * sizeof(int), hipMemcpyDeviceToHost));

    gettimeofday(&end, NULL); // stop the timer and calculate the Dijkstra execution time
    timersub(&end, &start, &dijkstra_time);

    //calculate all time program time
    gettimeofday(&all_end, NULL);
    timersub(&all_end, &all_start, &total_time);

    //print results
    FILE *fp;
    char filepath[200];
    sprintf(filepath, "ResultsOMP+CUDA/ResultsDijkstra/opt%d/type%d/OpenMP+CUDA_%d_%d_%d_%d.txt", atoi(argv[4]), type, N, n_edges, omp_get_max_threads(), numThreadsPerBlock); // define the path where to store the results
    char *filename = filepath;
    fp = fopen(filename, "w"); // open a file to write the results founded by the algorithm on it
    if (fp == NULL) {
        fprintf(stderr, "Error opening file %s for writing\n", filename);
        return 1; // Return an error code
    }
    fprintf(fp, "Distance Vector: \n");
    print_vector_on_file(fp,N,host_distances);
    fprintf(fp, "------------------------------------------------------\n");
    fprintf(fp, "Path Vector: \n");
    print_vector_on_file(fp,N,host_path);
    fprintf(fp, "------------------------------------------------------\n");
    fclose(fp);

    // Open a file to write al the calculated times founded by the algorithm
    FILE *fp2;
    char filepath2[200];
    sprintf(filepath2, "ResultsOMP+CUDA/InfoTimeDijkstra/opt%d/type%d/%d_%d.csv", atoi(argv[4]), type, N, n_edges); // define the path where to store the times calculated for each phase.
    char *filename2 = filepath2;
    fp2 = fopen(filename2, "a+");
    if (fp2 == NULL) {
        fprintf(stderr, "Error opening file %s for writing\n", filename2);
        return 1; // Return an error code
    }
    fprintf(fp2, "OpenMp+CUDA;%d;%d; %ld.%06ld; %ld.%06ld; %ld.%06ld; %ld.%06ld;\n", omp_get_max_threads(),numThreadsPerBlock,(long int)total_time.tv_sec, (long int)total_time.tv_usec, (long int)graph_creation_time.tv_sec, (long int)graph_creation_time.tv_usec, (long int)allocation_time.tv_sec, (long int)allocation_time.tv_usec,(long int)dijkstra_time.tv_sec, (long int)dijkstra_time.tv_usec);
    fclose(fp2);

    // Free device memory
    CUDA_SAFE_CALL(hipFree(device_graph));
    CUDA_SAFE_CALL(hipFree(device_distances));
    CUDA_SAFE_CALL(hipFree(device_path));
    CUDA_SAFE_CALL(hipFree(device_visited));

    // Free host memory
    freeGraph(graph);
    free(host_distances);
    free(host_path);
    free(host_visited);
    return 0;
}
